#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "sobel_cuda.h"

#define NUMBER_OF_BLOCKS 65535 
#define THREADS_PER_BLOCK 1024
#define FILTER_SIZE 9

__global__ void cu_filter_on_pic(int* original_image,int* filtered_image, 
    float* filter, int height, int width) {
  int surrounded_width = width + 2;
  int surrounded_height = height + 2;
  long image_length = surrounded_height * surrounded_width;

  int thread_id = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  if (thread_id < surrounded_width + 1 || 
      thread_id >= image_length - (surrounded_width + 1))
    return;

  if ((thread_id % surrounded_width) != 0 && 
      (thread_id % surrounded_width) != (surrounded_width - 1)) {
    float element = 
      original_image[thread_id - (surrounded_width + 1)] * filter[0] +
      original_image[thread_id - surrounded_width] * filter[1] + 
      original_image[thread_id - (surrounded_width - 1)] * filter[2] + 
      original_image[thread_id - 1] * filter[3] + 
      original_image[thread_id] * filter[4] + 
      original_image[thread_id + 1] * filter[5] + 
      original_image[thread_id + (surrounded_width - 1)] * filter[6] + 
      original_image[thread_id + surrounded_width] * filter[7] + 
      original_image[thread_id + (surrounded_width + 1)] * filter[8];

    if (element < 30)
      element = 0;
    element = (int)(round(element));
    long filtered_image_index = (thread_id - (surrounded_width + 1)) -
      (thread_id / surrounded_width - 1) * 2;
    filtered_image[filtered_image_index] = element;
  }
}

void filter_on_pic(int* original_image, int* filtered_image, 
    float* filter, int height, int width) {

  int* original_image_d;
  int* filtered_image_d;
  float* filter_d;
  int surrounded_width = width + 2;
  int surrounded_height = height + 2;
  int original_image_size = surrounded_width * surrounded_height;
  int filtered_image_size = height * width;
  hipError_t result_status;

  result_status = hipMalloc((void**) &original_image_d, 
      sizeof(int) * original_image_size);
  if (result_status != hipSuccess) {
    printf("hipMalloc - original_image_d - failed\n");
    exit(1);
  }

  result_status = hipMalloc((void**) &filtered_image_d, 
      sizeof(int) * filtered_image_size);
  if (result_status != hipSuccess) {
    printf("hipMalloc - filtered_image_d - failed\n");
    exit(1);
  }

  result_status = hipMalloc((void**) &filter_d, sizeof(float) * FILTER_SIZE);
  if (result_status != hipSuccess) {
    printf("hipMalloc - filter_d - failed\n");
    exit(1);
  }

  result_status = hipMemcpy(original_image_d, original_image, 
      sizeof(int) * original_image_size, hipMemcpyHostToDevice);
  if (result_status != hipSuccess) {
    printf("hipMemcpy - host-GPU - original_image - failed\n");
    exit(1);
  }

  result_status = hipMemcpy(filtered_image_d, filtered_image, 
      sizeof(int) * filtered_image_size, hipMemcpyHostToDevice);
  if (result_status != hipSuccess) {
    printf("hipMemcpy - host-GPU - filtered_image - failed\n");
    exit(1);
  }

  result_status = hipMemcpy(filter_d, filter, 
      sizeof(float) * FILTER_SIZE, hipMemcpyHostToDevice);
  if (result_status != hipSuccess) {
    printf("hipMemcpy - host-GPU - filtered_image - failed\n");
    exit(1);
  }

  dim3 dimblock(THREADS_PER_BLOCK);
  dim3 dimgrid(NUMBER_OF_BLOCKS);

  cu_filter_on_pic<<<dimgrid, dimblock>>>(original_image_d, filtered_image_d, 
      filter_d, height, width);

  result_status = hipMemcpy(filtered_image, filtered_image_d, 
      sizeof(int) * filtered_image_size, hipMemcpyDeviceToHost);
  if (result_status != hipSuccess) {
    printf("hipMemcpy - GPU-host - filtered_image_d - failed\n");
    exit(1);
  }

  hipFree(original_image_d);
  hipFree(filtered_image_d);
  hipFree(filter_d);
}
